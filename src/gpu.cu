#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
extern "C" {
    #include "gpu.h"
}

#define CONV_COL(l,c,nb_l) \
    (c)*(nb_l)+(l) 

__global__ void apply_blur_filter_one_iter_col_gpu( pixel * res, pixel * p, int * end, int width, int height, int size_stencil, int threshold){

    int n_blocks = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
    int n_thread_in_block = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
    
    int n = n_blocks * blockDim.x * blockDim.y * blockDim.z + n_thread_in_block;

    int n_pixels = width * height;

    int i = n % height;
    int j = n / height;
    int blurred_top = ( i >= size_stencil && i < height/10 && i >= size_stencil && i < width-size_stencil);
    int blurred_bottom = ( i < height-size_stencil && i >= height*9/10 && i >= size_stencil && i < n_blocks );
    int blurred = blurred_bottom || blurred_top;

    if( n < n_pixels){
        // 1. copy
        res[n].r = p[n].r;
        res[n].g = p[n].g;
        res[n].b = p[n].b;

        // 2. blur on top and bottom
        if ( blurred ){
            int stencil_i, stencil_j ;
            int t_r = 0 ;
            int t_g = 0 ;
            int t_b = 0 ;

            for ( stencil_i = -size_stencil ; stencil_i <= size_stencil ; stencil_i++ )
            {
                for ( stencil_j = -size_stencil ; stencil_j <= size_stencil ; stencil_j++ )
                {
                    t_r += p[CONV_COL(i+stencil_i, j+stencil_j,height)].r ;
                    t_g += p[CONV_COL(i+stencil_i, j+stencil_j,height)].g ;
                    t_b += p[CONV_COL(i+stencil_i, j+stencil_j,height)].b ;
                }
            }

            res[n].r = t_r / ( (2*size_stencil+1)*(2*size_stencil+1) ) ;
            res[n].g = t_g / ( (2*size_stencil+1)*(2*size_stencil+1) ) ;
            res[n].b = t_b / ( (2*size_stencil+1)*(2*size_stencil+1) ) ;
        } else {
            // Copy middle part fo image
            res[n].r = p[n].r;
            res[n].g = p[n].g;
            res[n].b = p[n].b;
        }

        float diff_r ;
        float diff_g ;
        float diff_b ;

        diff_r = res[n].r - p[n].r ;
        diff_g = res[n].g - p[n].g ;
        diff_b = res[n].b - p[n].b ;

        if ( diff_r > threshold || -diff_r > threshold 
                ||
                    diff_g > threshold || -diff_g > threshold
                    ||
                    diff_b > threshold || -diff_b > threshold
            ) {
            *end = 0 ;
        }

        p[n].r = res[n].r ;
        p[n].g = res[n].g ;
        p[n].b = res[n].b ;
    }
}

// img in COLUMNS
extern "C"
void gpu_part(int width, int height, pixel *p, int size, int threshold, pixel *res, int *end)
{
    int length = width * height ;
    int *d_end;
    pixel *d_p, *d_res;
    hipError_t err;

    dim3 bl, t;
    int n_threads = 1000;
    int n_blocks = length / n_threads;

    if(( err = hipMalloc((void **)&d_p, length * sizeof(pixel)) ) != hipSuccess)
        printf("\tERROR when malloc 1 : %s\n", hipGetErrorString(err));
    if( (err = hipMalloc((void **)&d_res, length * sizeof(pixel)) ) != hipSuccess)
        printf("\tERROR when malloc 2: %s\n", hipGetErrorString(err));
    if( (err = hipMalloc((void **)&d_end, sizeof(int)) ) != hipSuccess)
        printf("\tERROR when malloc 3: %s\n", hipGetErrorString(err));
    if( (err = hipMemcpy(d_p, p, length * sizeof(pixel), hipMemcpyHostToDevice) ) != hipSuccess)
        printf("\tERROR when copy 1: %s\n", hipGetErrorString(err));
    if( (err = hipMemcpy(d_end, end, sizeof(int), hipMemcpyHostToDevice) ) != hipSuccess)
        printf("\tERROR when copy 2: %s\n", hipGetErrorString(err));   

    bl.x = n_blocks ;
    bl.y = 1 ;
    bl.z = 1 ;

    t.x = n_threads ;
    t.y = 1 ;

    apply_blur_filter_one_iter_col_gpu<<<bl,t>>>( d_res, d_p, end, width, height, 5, threshold ) ;
    hipDeviceSynchronize();
    if( (err = hipMemcpy(end, d_end, sizeof(int), hipMemcpyDeviceToHost) ) != hipSuccess)
        printf("\tERROR when copy 3: %s\n", hipGetErrorString(err));   
    
    hipDeviceSynchronize();
    printf("\tEND : %d\n", *end);
    // hipMemcpy(res, d_res, length * sizeof(pixel), hipMemcpyDeviceToHost);
}