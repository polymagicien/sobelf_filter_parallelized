#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
extern "C" {
    #include "gpu.h"
}

#define CONV_COL(l,c,nb_l) \
    (c)*(nb_l)+(l) 

#define DIV 3

__global__ void apply_blur_filter_one_iter_col_gpu( pixel * res, pixel * p, int * end, int width, int height, int size_stencil, int threshold){
  
    int n = blockIdx.x * blockDim.x  + threadIdx.x;
    int n_pixels = width * height;

    int i = n % height;
    int j = n / height;

    int blurred_top = ( i >= size_stencil && i < height/10-size_stencil  && j >= size_stencil && j < width-size_stencil);
    int blurred_bottom = ( i < height-size_stencil && i >= height*0.9+size_stencil && j >= size_stencil && j < width-size_stencil );
    int blurred = blurred_bottom || blurred_top;

    if( n < n_pixels){
        // 1. copy
        res[n].r = p[n].r;
        res[n].g = p[n].g;
        res[n].b = p[n].b;

        // 2. blur on top and bottom
        if ( blurred ){
            int stencil_i, stencil_j ;
            int t_r = 0 ;
            int t_g = 0 ;
            int t_b = 0 ;

            for ( stencil_i = -size_stencil ; stencil_i <= size_stencil ; stencil_i++ )
            {
                for ( stencil_j = -size_stencil ; stencil_j <= size_stencil ; stencil_j++ )
                {
                    t_r += p[CONV_COL(i+stencil_i, j+stencil_j,height)].r ;
                    t_g += p[CONV_COL(i+stencil_i, j+stencil_j,height)].g ;
                    t_b += p[CONV_COL(i+stencil_i, j+stencil_j,height)].b ;
                }
            }

            res[n].r = t_r / ( (2*size_stencil+1)*(2*size_stencil+1) ) ;
            res[n].g = t_g / ( (2*size_stencil+1)*(2*size_stencil+1) ) ;
            res[n].b = t_b / ( (2*size_stencil+1)*(2*size_stencil+1) ) ;
        } else {
            // Copy middle part fo image
            res[n].r = p[n].r;
            res[n].g = p[n].g;
            res[n].b = p[n].b;
        }

        float diff_r ;
        float diff_g ;
        float diff_b ;

        diff_r = res[n].r - p[n].r ;
        diff_g = res[n].g - p[n].g ;
        diff_b = res[n].b - p[n].b ;

        if ( diff_r > threshold || -diff_r > threshold 
                ||
                    diff_g > threshold || -diff_g > threshold
                    ||
                    diff_b > threshold || -diff_b > threshold
            ) {
            *end = 0 ;
        }
    }
}

// img in COLUMNS
extern "C"
void gpu_part(int width, int height, pixel *p, int size, int threshold, pixel *res, int *end)
{
    int length = width * height ;
    int *d_end;
    pixel *d_p, *d_res;
    hipError_t err;

    dim3 bl, t;
    int n_threads = 1000;
    int n_blocks = length / n_threads + 1;

    if(( err = hipMalloc((void **)&d_p, length * sizeof(pixel)) ) != hipSuccess)
        printf("\tERROR when malloc 1 : %s\n", hipGetErrorString(err));
    if( (err = hipMalloc((void **)&d_res, length * sizeof(pixel)) ) != hipSuccess)
        printf("\tERROR when malloc 2: %s\n", hipGetErrorString(err));
    if( (err = hipMalloc((void **)&d_end, sizeof(int)) ) != hipSuccess)
        printf("\tERROR when malloc 3: %s\n", hipGetErrorString(err));
    if( (err = hipMemcpy(d_p, p, length * sizeof(pixel), hipMemcpyHostToDevice) ) != hipSuccess)
        printf("\tERROR when copy 1: %s\n", hipGetErrorString(err));
    if( (err = hipMemcpy(d_end, end, sizeof(int), hipMemcpyHostToDevice) ) != hipSuccess)
        printf("\tERROR when copy 2: %s\n", hipGetErrorString(err));   

    bl.x = n_blocks ;
    t.x = n_threads ;

    apply_blur_filter_one_iter_col_gpu<<<bl,t>>>( d_res, d_p, d_end, width, height, size, threshold ) ;
    hipDeviceSynchronize();

    if( (err = hipMemcpy(end, d_end, sizeof(int), hipMemcpyDeviceToHost) ) != hipSuccess)
        printf("\tERROR when copy 3: %s\n", hipGetErrorString(err));   
    hipDeviceSynchronize();

    if( (err = hipMemcpy(p, d_res, length * sizeof(pixel), hipMemcpyDeviceToHost) ) != hipSuccess)
        printf("\tERROR when copy 4: %s\n", hipGetErrorString(err));   
    hipDeviceSynchronize();

    hipFree(d_p);
    hipFree(d_res);
    hipFree(d_end);
}